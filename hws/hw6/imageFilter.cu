
#include <hip/hip_runtime.h>
#include <stdio.h>

// A macro for checking the error codes of cuda runtime calls
#define CUDA_ERROR_CHECK(expr) \
  {                            \
    hipError_t err = expr;    \
    if (err != hipSuccess)    \
    {                          \
      printf("CUDA call failed!\n%s\n", hipGetErrorString(err)); \
      exit(1);                 \
    }                          \
  }


__global__
void swapChannel_kernel(uchar3 * device_inputImage, uchar3 * device_outputImage, int rows, int cols)
{
  //----------------------------------------------------------------
  // TODO: Implement this Kernel
  //----------------------------------------------------------------

}

__global__
void blurImage_kernel(uchar3 * device_inputImage, uchar3 * device_outputImage, int rows, int cols)
{
  //----------------------------------------------------------------
  // TODO: Implement this Kernel
  //----------------------------------------------------------------
}

__global__
void inplaceFlip_kernel(uchar3 * device_outputImage, int rows, int cols)
{
  //----------------------------------------------------------------
  // TODO: Implement this Kernel
  //----------------------------------------------------------------

}

__global__
void creative_kernel(uchar3 * device_inputImage, uchar3 * device_outputImage, int rows, int cols)
{
  //----------------------------------------------------------------
  // TODO: Implement this Kernel
  //----------------------------------------------------------------

}


__host__
float filterImage(uchar3 *host_inputImage, uchar3 *host_outputImage, int rows, int cols, int filterNumber){

  int numPixels = rows * cols;

  //allocate memory on device (GPU)
  uchar3 *device_inputImage;
  uchar3 *device_outputImage;

  CUDA_ERROR_CHECK(hipMalloc(&device_inputImage, sizeof(uchar3) * numPixels));
  CUDA_ERROR_CHECK(hipMalloc(&device_outputImage, sizeof(uchar3) * numPixels));
  CUDA_ERROR_CHECK(hipMemset(device_outputImage, 0,  sizeof(uchar3) * numPixels)); //make sure no memory is left laying around

  //copy input image to the device (GPU)
  CUDA_ERROR_CHECK(hipMemcpy(device_inputImage, host_inputImage, sizeof(uchar3) * numPixels, hipMemcpyHostToDevice));

  //start timing to measure length of kernel call
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  //----------------------------------------------------------------
  // TODO: Fill in the parameters for the kernel calls
  //----------------------------------------------------------------
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)

  // Also note that you pass the pointers to the device memory to the kernel call

  int gridSize = 1;
  int blockSize = 1;

  switch(filterNumber){
    case 1:
      swapChannel_kernel<<<gridSize,blockSize>>>(device_inputImage, device_outputImage, rows, cols);
      break;
    case 2:
      blurImage_kernel<<<gridSize,blockSize>>>(device_inputImage, device_outputImage, rows, cols);
      break;
    case 3:
      inplaceFlip_kernel<<<gridSize,blockSize>>>(device_inputImage, rows, cols);
      break;
    case 4:
      creative_kernel<<<gridSize,blockSize>>>(device_inputImage, device_outputImage, rows, cols);
      break;
    default:
      break;
  }

  //----------------------------------------------------------------
  // END KERNEL CALLS - Do not modify code beyond this point!
  //----------------------------------------------------------------

  //stop timing
  hipEventRecord(stop);

  hipEventSynchronize(stop);

  float timeElapsedInMs = 0;
  hipEventElapsedTime(&timeElapsedInMs, start, stop);

  //synchronize
  hipDeviceSynchronize(); CUDA_ERROR_CHECK(hipGetLastError());

  //copy device output image back to host output image
  //special case for filter swap - since it is in place, we actually copy the input image back to the host output
  if (filterNumber==3){
    CUDA_ERROR_CHECK(hipMemcpy(host_outputImage, device_inputImage, sizeof(uchar3) * numPixels, hipMemcpyDeviceToHost));
  }else{
    CUDA_ERROR_CHECK(hipMemcpy(host_outputImage, device_outputImage, sizeof(uchar3) * numPixels, hipMemcpyDeviceToHost));
  }


  //free Memory
  CUDA_ERROR_CHECK(hipFree(device_inputImage));
  CUDA_ERROR_CHECK(hipFree(device_outputImage));

  return timeElapsedInMs;
}
